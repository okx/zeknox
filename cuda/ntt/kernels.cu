#include "hip/hip_runtime.h"
// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#ifndef __CRYPTO_KERNELS_CU__
#define __CRYPTO_KERNELS_CU__
#include <hip/hip_cooperative_groups.h>

/**
 * \param  i, the integer to be bit reversed, i is in range [0, 1<<nbits)
 * \param nbits, number of bits to represent the integer. e.g Goldilocks Field, log_n_size <=32, most of the time, nbits is <= 32
 * __brev will treat i as 32 bits integer and return the bit reversed integer, whose least (32 - nbits) bits are all zero.
 */
__device__ __forceinline__
    index_t
    bit_rev(index_t i, unsigned int nbits)
{
    if (sizeof(i) == 4 || nbits <= 32)
        return __brev(i) >> (8 * sizeof(unsigned int) - nbits);
    else
        return __brevll(i) >> (8 * sizeof(unsigned long long) - nbits);
}

#ifdef __CUDA_ARCH__
__device__ __forceinline__ void shfl_bfly(fr_t &r, int laneMask)
{
#pragma unroll
    for (int iter = 0; iter < r.len(); iter++)
        r[iter] = __shfl_xor_sync(0xFFFFFFFF, r[iter], laneMask);
}
#endif

__device__ __forceinline__ void shfl_bfly(index_t &index, int laneMask)
{
    index = __shfl_xor_sync(0xFFFFFFFF, index, laneMask);
}

// Permutes the data in an array such that data[i] = data[bit_reverse(i)]
// and data[bit_reverse(i)] = data[i]
__launch_bounds__(1024) __global__
    void bit_rev_permutation(fr_t *d_out, const fr_t *d_in, uint32_t lg_domain_size)
{

    index_t i = threadIdx.x + blockDim.x * (index_t)blockIdx.x;
    index_t r = bit_rev(i, lg_domain_size);
    // printf("invoking bit_rev_permutation i: %d, r: %d, lg_domain_size: %d\n", i, r, lg_domain_size);
    if (i < r || (d_out != d_in && i == r))
    { // if r=i, no need to swap; if i<r, swap data at i and r;
        fr_t t0 = d_in[i];
        fr_t t1 = d_in[r];
        d_out[r] = t0;
        d_out[i] = t1;
        // printf("invoking bit_rev_permutation r: %d, t0: %lu \n", r, t0);
        // printf("invoking bit_rev_permutation i: %d, t1: %lu \n", i, t1);
    }
}

template <typename T>
static __device__ __host__ constexpr uint32_t lg2(T n)
{
    uint32_t ret = 0;
    while (n >>= 1)
        ret++;
    return ret;
}

/** for goldilocks field
 * if lg_domain_size=14, #blocks = 4, blockDim = 128; 4*128*32 = 1<< (2+5+7) = 1<<14
 */
__global__ void bit_rev_permutation_aux(fr_t *out, const fr_t *in, uint32_t lg_domain_size)
{
    const size_t Z_COUNT = 256 / sizeof(fr_t); // 32 for goldilocks field
    const uint32_t LG_Z_COUNT = lg2(Z_COUNT);  // 5 for goldilocks

    extern __shared__ fr_t exchange[]; // dynamically allocated shared memory within a CUDA kernel. Shared memory is a type of memory that is shared among threads within the same thread block and resides on-chip
    fr_t(*xchg)[Z_COUNT][Z_COUNT] = reinterpret_cast<decltype(xchg)>(exchange);

    index_t step = (index_t)1 << (lg_domain_size - LG_Z_COUNT);                         // if lg_domain_size = 14, it is 1<<9; treat all blocks (and threads) as the column size, 32 is row size
    index_t group_idx = (threadIdx.x + blockDim.x * (index_t)blockIdx.x) >> LG_Z_COUNT; // col index divided by Z_COUNT, range [[0]*32,[1]*32,[2]*32,[3]*32, ... [15]*32]
    uint32_t brev_limit = lg_domain_size - LG_Z_COUNT * 2;
    index_t brev_mask = ((index_t)1 << brev_limit) - 1;
    index_t group_idx_brev =
        (group_idx & ~brev_mask) | bit_rev(group_idx & brev_mask, brev_limit);
    uint32_t group_thread = threadIdx.x & (Z_COUNT - 1); // group_thread in range [0..32] * 4
    uint32_t group_thread_rev = bit_rev(group_thread, LG_Z_COUNT);
    uint32_t group_in_block_idx = threadIdx.x >> LG_Z_COUNT; // group_in_block_idx in range [[0]*32,[1]*32,[2]*32,[3]*32]

#pragma unroll
    for (uint32_t i = 0; i < Z_COUNT; i++)
    {
        xchg[group_in_block_idx][i][group_thread_rev] =
            in[group_idx * Z_COUNT + i * step + group_thread];
    }

    if (Z_COUNT > WARP_SZ)
        __syncthreads(); // is used to synchronize threads within the same block, ensuring that all shared memory writes are visible to all threads before proceeding with further computation
    else
        __syncwarp();

#pragma unroll
    for (uint32_t i = 0; i < Z_COUNT; i++)
    {
        out[group_idx_brev * Z_COUNT + i * step + group_thread] =
            xchg[group_in_block_idx][group_thread_rev][i];
    }
}

__device__ __forceinline__
    fr_t
    get_intermediate_root(index_t pow, const fr_t (*roots)[WINDOW_SIZE],
                          unsigned int nbits = MAX_LG_DOMAIN_SIZE)
{
    unsigned int off = 0;

    fr_t t, root = roots[off][pow % WINDOW_SIZE];
#pragma unroll 1
    while (pow >>= LG_WINDOW_SIZE)
        root *= (t = roots[++off][pow % WINDOW_SIZE]);

    return root;
}

__device__ __forceinline__ void get_intermediate_roots(fr_t &root0, fr_t &root1,
                                                       index_t idx0, index_t idx1,
                                                       const fr_t (*roots)[WINDOW_SIZE])
{
    int win = (WINDOW_NUM - 1) * LG_WINDOW_SIZE;
    int off = (WINDOW_NUM - 1);

    root0 = roots[off][idx0 >> win];
    root1 = roots[off][idx1 >> win];
#pragma unroll 1
    while (off--)
    {
        fr_t t;
        win -= LG_WINDOW_SIZE;
        root0 *= (t = roots[off][(idx0 >> win) % WINDOW_SIZE]);
        root1 *= (t = roots[off][(idx1 >> win) % WINDOW_SIZE]);
    }
}

template <unsigned int z_count>
__device__ __forceinline__ void coalesced_load(fr_t r[z_count], const fr_t *inout, index_t idx,
                                               const unsigned int stage)
{
    const unsigned int x = threadIdx.x & (z_count - 1);
    idx &= ~((index_t)(z_count - 1) << stage);
    idx += x;

#pragma unroll
    for (int z = 0; z < z_count; z++, idx += (index_t)1 << stage)
        r[z] = inout[idx];
}

template <unsigned int z_count>
__device__ __forceinline__ void transpose(fr_t r[z_count])
{
    extern __shared__ fr_t shared_exchange[];
    fr_t(*xchg)[z_count] = reinterpret_cast<decltype(xchg)>(shared_exchange);

    const unsigned int x = threadIdx.x & (z_count - 1);
    const unsigned int y = threadIdx.x & ~(z_count - 1);

#pragma unroll
    for (int z = 0; z < z_count; z++)
        xchg[y + z][x] = r[z];

    __syncwarp();

#pragma unroll
    for (int z = 0; z < z_count; z++)
        r[z] = xchg[y + x][z];
}

template <unsigned int z_count>
__device__ __forceinline__ void coalesced_store(fr_t *inout, index_t idx, const fr_t r[z_count],
                                                const unsigned int stage)
{
    const unsigned int x = threadIdx.x & (z_count - 1);
    idx &= ~((index_t)(z_count - 1) << stage);
    idx += x;

#pragma unroll
    for (int z = 0; z < z_count; z++, idx += (index_t)1 << stage)
        inout[idx] = r[z];
}

#if defined(FEATURE_GOLDILOCKS)
const static int Z_COUNT = 256 / 8 / sizeof(fr_t);
#include "kernels/ct_mixed_radix_narrow.cu"
#else // 256-bit fields
#include "kernels/ct_mixed_radix_wide.cu"
#endif

#include <util/sharedmem.cuh>
__global__ void reverse_order_kernel(fr_t *arr, fr_t *arr_reversed, uint32_t n, uint32_t logn, uint32_t batch_size)
{
    int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (threadId < n * batch_size)
    {
        int idx = threadId % n;
        int batch_idx = threadId / n;
        int idx_reversed = __brev(idx) >> (32 - logn);
        arr_reversed[batch_idx * n + idx_reversed] = arr[batch_idx * n + idx];
    }
}

/**
 * Cooley-Tuckey NTT.
 * NOTE! this function assumes that d_twiddles are located in the device memory.
 * @param arr_in input array of type E (elements).
 * @param n length of d_arr.
 * @param twiddles twiddle factors of type S (scalars) array allocated on the device memory (must be a power of 2).
 * @param n_twiddles length of twiddles, should be negative for intt.
 * @param max_task max count of parallel tasks.
 * @param s log2(n) loop index.
 * @param arr_out buffer for the output.
 */
__global__ void ntt_template_kernel_shared(
    fr_t *__restrict__ arr_in,
    int n,
    const fr_t *__restrict__ r_twiddles,
    int n_twiddles,
    int max_task,
    int s,
    int logn,
    fr_t *__restrict__ arr_out)
{
    SharedMemory<fr_t> smem;
    fr_t *arr = smem.getPointer();

    uint32_t task = blockIdx.x;
    uint32_t loop_limit = blockDim.x;
    uint32_t chunks = n / (loop_limit * 2);
    uint32_t offset = (task / chunks) * n;
    if (task < max_task)
    {
        // flattened loop allows parallel processing
        uint32_t l = threadIdx.x;

        if (l < loop_limit)
        {
#pragma unroll
            for (; s < logn; s++) // TODO: this loop also can be unrolled
            {
                uint32_t ntw_i = task % chunks;

                uint32_t n_twiddles_div = n_twiddles >> (s + 1);

                uint32_t shift_s = 1 << s;
                uint32_t shift2_s = 1 << (s + 1);

                l = ntw_i * loop_limit + l; // to l from chunks to full

                uint32_t j = l & (shift_s - 1);               // Equivalent to: l % (1 << s)
                uint32_t i = ((l >> s) * shift2_s) & (n - 1); // (..) % n (assuming n is power of 2)
                uint32_t oij = i + j;
                uint32_t k = oij + shift_s;
                fr_t tw = *(r_twiddles + (int)(j * n_twiddles_div));

                fr_t u = s == 0 ? arr_in[offset + oij] : arr[oij];
                fr_t v = s == 0 ? arr_in[offset + k] : arr[k];
                v = tw * v;
                if (s == (logn - 1))
                {
                    arr_out[offset + oij] = u + v;
                    arr_out[offset + k] = u - v;
                }
                else
                {
                    arr[oij] = u + v;
                    arr[k] = u - v;
                }

                __syncthreads();
            }
        }
    }
}

/**
 * Cooley-Tukey NTT.
 * NOTE! this function assumes that d_twiddles are located in the device memory.
 * @param arr input array of type E (elements).
 * @param n length of d_arr.
 * @param twiddles twiddle factors of type S (scalars) array allocated on the device memory (must be a power of 2).
 * @param n_twiddles length of twiddles, should be negative for intt.
 * @param max_task max count of parallel tasks.
 * @param s log2(n) loop index.
 */
__global__ void
ntt_template_kernel(fr_t *arr_in, int n, fr_t *twiddles, int n_twiddles, int max_task, int s, bool rev, fr_t *arr_out)
{
    int task = blockIdx.x;
    int chunks = n / (blockDim.x * 2);

    if (task < max_task)
    {
        // flattened loop allows parallel processing
        uint32_t l = threadIdx.x;
        uint32_t loop_limit = blockDim.x;

        if (l < loop_limit)
        {
            uint32_t ntw_i = task % chunks;

            uint32_t shift_s = 1 << s;
            uint32_t shift2_s = 1 << (s + 1);
            uint32_t n_twiddles_div = n_twiddles >> (s + 1);

            l = ntw_i * blockDim.x + l; // to l from chunks to full

            uint32_t j = l & (shift_s - 1);               // Equivalent to: l % (1 << s)
            uint32_t i = ((l >> s) * shift2_s) & (n - 1); // (..) % n (assuming n is power of 2)
            uint32_t k = i + j + shift_s;

            fr_t tw = *(twiddles + (int)(j * n_twiddles_div));

            uint32_t offset = (task / chunks) * n;
            fr_t u = arr_in[offset + i + j];
            fr_t v = arr_in[offset + k];
            if (!rev)
                v = tw * v;
            arr_out[offset + i + j] = u + v;
            v = u - v;
            if (rev)
            {
                arr_out[offset + k] = ((fr_t)tw * v);
            }
            else
            {
                arr_out[offset + k] = v;
            }
            //  = rev ? : (fr_t)v;
        }
    }
}

 /**
     * Cooley-Tuckey NTT.
     * NOTE! this function assumes that d_twiddles are located in the device memory.
     * @param arr_in input array of type E (elements).
     * @param n length of d_arr.
     * @param twiddles twiddle factors of type S (scalars) array allocated on the device memory (must be a power of 2).
     * @param n_twiddles length of twiddles, should be negative for intt.
     * @param max_task max count of parallel tasks.
     * @param s log2(n) loop index.
     * @param arr_out buffer for the output.
     */
    __global__ void ntt_template_kernel_shared_rev(
      fr_t* __restrict__ arr_in,
      int n,
      const fr_t* __restrict__ r_twiddles,
      int n_twiddles,
      int max_task,
      int ss,
      int logn,
      fr_t* __restrict__ arr_out)
    {
      SharedMemory<fr_t> smem;
      fr_t* arr = smem.getPointer();

      uint32_t task = blockIdx.x;
      uint32_t loop_limit = blockDim.x;
      uint32_t chunks = n / (loop_limit * 2);
      uint32_t offset = (task / chunks) * n;
      if (task < max_task) {
        // flattened loop allows parallel processing
        uint32_t l = threadIdx.x;

        if (l < loop_limit) {
#pragma unroll
          for (; ss < logn; ss++) {
            int s = logn - ss - 1;
            bool is_beginning = ss == 0;
            bool is_end = ss == (logn - 1);

            uint32_t ntw_i = task % chunks;

            uint32_t n_twiddles_div = n_twiddles >> (s + 1);

            uint32_t shift_s = 1 << s;
            uint32_t shift2_s = 1 << (s + 1);

            l = ntw_i * loop_limit + l; // to l from chunks to full

            uint32_t j = l & (shift_s - 1);               // Equivalent to: l % (1 << s)
            uint32_t i = ((l >> s) * shift2_s) & (n - 1); // (..) % n (assuming n is power of 2)
            uint32_t oij = i + j;
            uint32_t k = oij + shift_s;

            fr_t tw = *(r_twiddles + (int)(j * n_twiddles_div));

            fr_t u = is_beginning ? arr_in[offset + oij] : arr[oij];
            fr_t v = is_beginning ? arr_in[offset + k] : arr[k];
            if (is_end) {
              arr_out[offset + oij] = u + v;
              arr_out[offset + k] = tw * (u - v);
            } else {
              arr[oij] = u + v;
              arr[k] = tw * (u - v);
            }

            __syncthreads();
          }
        }
      }
    }


#endif /**__CRYPTO_KERNELS_CU__ */