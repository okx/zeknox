#include "hip/hip_runtime.h"
// sha3.c
// 19-Nov-11  Markku-Juhani O. Saarinen <mjos@iki.fi>

// Revised 07-Aug-15 to match with official release of FIPS PUB 202 "SHA3"
// Revised 03-Sep-15 for portability + OpenSSL - style API

#include <stdio.h>
#include <stdint.h>
#include "cuda_util.h"

#define TPB 128
#define MAX_RECORD  2048    // max 2048 bytes

#ifndef KECCAKF_ROUNDS
#define KECCAKF_ROUNDS 24
#endif

#ifndef ROTL64
#define ROTL64(x, y) (((x) << (y)) | ((x) >> (64 - (y))))
#endif

// state context
typedef struct
{
    union
    {                   // state:
        uint8_t b[200]; // 8-bit bytes
        uint64_t q[25]; // 64-bit words
    } st;
    int pt, rsiz, mdlen; // these don't overflow
} sha3_ctx_t;

// constants
const uint64_t host_keccakf_rndc[24] = {
    0x0000000000000001, 0x0000000000008082, 0x800000000000808a,
    0x8000000080008000, 0x000000000000808b, 0x0000000080000001,
    0x8000000080008081, 0x8000000000008009, 0x000000000000008a,
    0x0000000000000088, 0x0000000080008009, 0x000000008000000a,
    0x000000008000808b, 0x800000000000008b, 0x8000000000008089,
    0x8000000000008003, 0x8000000000008002, 0x8000000000000080,
    0x000000000000800a, 0x800000008000000a, 0x8000000080008081,
    0x8000000000008080, 0x0000000080000001, 0x8000000080008008};
const int host_keccakf_rotc[24] = {
    1, 3, 6, 10, 15, 21, 28, 36, 45, 55, 2, 14,
    27, 41, 56, 8, 25, 43, 62, 18, 39, 61, 20, 44};
const int host_keccakf_piln[24] = {
    10, 7, 11, 17, 18, 3, 5, 16, 8, 21, 24, 4,
    15, 23, 19, 13, 12, 2, 20, 14, 22, 9, 6, 1};

__device__ __constant__ uint64_t gpu_keccakf_rndc[24];
__device__ __constant__ int gpu_keccakf_rotc[24];
__device__ __constant__ int gpu_keccakf_piln[24];

sha3_ctx_t *gpu_contexts;
char *gpu_in;
int *gpu_inlen;
int *gpu_offset;
char *gpu_md;

int *offsets;
char *out;

// update the state with given number of rounds

__device__ void sha3_keccakf(uint64_t st[25])
{
    // variables
    int i, j, r;
    uint64_t t, bc[5];

#if __BYTE_ORDER__ != __ORDER_LITTLE_ENDIAN__
    uint8_t *v;

    // endianess conversion. this is redundant on little-endian targets
    for (i = 0; i < 25; i++)
    {
        v = (uint8_t *)&st[i];
        st[i] = ((uint64_t)v[0]) | (((uint64_t)v[1]) << 8) |
                (((uint64_t)v[2]) << 16) | (((uint64_t)v[3]) << 24) |
                (((uint64_t)v[4]) << 32) | (((uint64_t)v[5]) << 40) |
                (((uint64_t)v[6]) << 48) | (((uint64_t)v[7]) << 56);
    }
#endif

    // actual iteration
    for (r = 0; r < KECCAKF_ROUNDS; r++)
    {

        // Theta
        for (i = 0; i < 5; i++)
            bc[i] = st[i] ^ st[i + 5] ^ st[i + 10] ^ st[i + 15] ^ st[i + 20];

        for (i = 0; i < 5; i++)
        {
            t = bc[(i + 4) % 5] ^ ROTL64(bc[(i + 1) % 5], 1);
            for (j = 0; j < 25; j += 5)
                st[j + i] ^= t;
        }

        // Rho Pi
        t = st[1];
        for (i = 0; i < 24; i++)
        {
            j = gpu_keccakf_piln[i];
            bc[0] = st[j];
            st[j] = ROTL64(t, gpu_keccakf_rotc[i]);
            t = bc[0];
        }

        //  Chi
        for (j = 0; j < 25; j += 5)
        {
            for (i = 0; i < 5; i++)
                bc[i] = st[j + i];
            for (i = 0; i < 5; i++)
                st[j + i] ^= (~bc[(i + 1) % 5]) & bc[(i + 2) % 5];
        }

        //  Iota
        st[0] ^= gpu_keccakf_rndc[r];
    }

#if __BYTE_ORDER__ != __ORDER_LITTLE_ENDIAN__
    // endianess conversion. this is redundant on little-endian targets
    for (i = 0; i < 25; i++)
    {
        v = (uint8_t *)&st[i];
        t = st[i];
        v[0] = t & 0xFF;
        v[1] = (t >> 8) & 0xFF;
        v[2] = (t >> 16) & 0xFF;
        v[3] = (t >> 24) & 0xFF;
        v[4] = (t >> 32) & 0xFF;
        v[5] = (t >> 40) & 0xFF;
        v[6] = (t >> 48) & 0xFF;
        v[7] = (t >> 56) & 0xFF;
    }
#endif
}

// Initialize the context for SHA3

__device__ void sha3_init(sha3_ctx_t *c, int mdlen)
{
    int i;

    for (i = 0; i < 25; i++)
        c->st.q[i] = 0;
    c->mdlen = mdlen;
    c->rsiz = 200 - 2 * mdlen;
    c->pt = 0;
}

// update state with more data

__device__ void sha3_update(sha3_ctx_t *c, const void *data, size_t len)
{
    size_t i;
    int j;

    j = c->pt;
    for (i = 0; i < len; i++)
    {
        c->st.b[j++] ^= ((const uint8_t *)data)[i];
        if (j >= c->rsiz)
        {
            sha3_keccakf(c->st.q);
            j = 0;
        }
    }
    c->pt = j;
}

// finalize and output a hash
__device__ void sha3_final(void *md, sha3_ctx_t *c)
{
    int i;

    c->st.b[c->pt] ^= 0x06;
    c->st.b[c->rsiz - 1] ^= 0x80;
    sha3_keccakf(c->st.q);

    for (i = 0; i < c->mdlen; i++)
    {
        ((uint8_t *)md)[i] = c->st.b[i];
    }
}

// compute a SHA-3 hash (md) of given byte length from "in"

__global__ void sha3(int num, sha3_ctx_t *contexts, const void *in, int *inlen, int *offset, void *md, int mdlen)
{
    // int const tid = threadIdx.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num)
        return;

    sha3_init(contexts + tid, mdlen);
    sha3_update(contexts + tid, in + offset[tid], inlen[tid]);
    sha3_final(md + (tid * mdlen), contexts + tid);
}
