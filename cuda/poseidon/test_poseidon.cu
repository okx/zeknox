#include "hip/hip_runtime.h"
#include "poseidon.cuh"
#include "poseidon.h"

#include <stdio.h>

void printhash(u64 *h)
{
    for (int i = 0; i < 4; i++)
    {
        printf("%lu ", h[i]);
    }
    printf("\n");
}

__global__ void hash(uint64_t *in, uint64_t *out, uint32_t n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid > 0)
        return;

    gpu_poseidon_hash_one((gl64_t *)in, n, (gl64_t *)out);
}

__global__ void hash_step1(uint64_t *in, uint64_t *out, uint32_t n, uint32_t len)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= len)
        return;

    gpu_poseidon_hash_one((gl64_t *)(in + n * tid), n, (gl64_t *)(out + 4 * tid));
}

__global__ void hash_step2(uint64_t *in, uint64_t *out, uint32_t len)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= len)
        return;

    gpu_poseidon_hash_two((gl64_t *)(in + 8 * tid), (gl64_t *)(in + 8 * tid + 4), (gl64_t *)(out + 4 * tid));
}

int test1()
{
    u64 leaf[7] = {8395359103262935841, 1377884553022145855, 2370707998790318766, 3651132590097252162, 1141848076261006345, 12736915248278257710, 9898074228282442027};

    u64 h1[4] = {0u};
    u64 h2[4] = {0u};

    u64 *gpu_leaf;
    u64 *gpu_hash;
    CHECKCUDAERR(hipMalloc(&gpu_leaf, 6 * sizeof(u64)));
    CHECKCUDAERR(hipMalloc(&gpu_hash, 4 * sizeof(u64)));
    CHECKCUDAERR(hipMemcpy(gpu_leaf, leaf, 6 * sizeof(u64), hipMemcpyHostToDevice));

    for (int k = 2; k <= 6; k += 2)
    {
        hash<<<1, 1>>>(gpu_leaf, gpu_hash, k);
        CHECKCUDAERR(hipMemcpy(h1, gpu_hash, 4 * sizeof(u64), hipMemcpyDeviceToHost));
        printhash(h1);
        cpu_poseidon_hash_one(leaf, k, h2);
        printhash(h2);

        for (int j = 0; j < 4; j++) {
            if (h1[j] != h2[j]) {
                printf("ERROR: CPU and GPU results are different!\n");
                break;
            }
        }
    }

    /*
    #ifdef RUST_POSEIDON
        ext_poseidon_hash_or_noop(h1, leaf, 1);
        printhash(h1);
    #endif
        cpu_poseidon_hash_one(leaf, 1, h2);
        printhash(h2);

    #ifdef RUST_POSEIDON
        ext_poseidon_hash_or_noop(h1, leaf, 4);
        printhash(h1);
    #endif
        cpu_poseidon_hash_one(leaf, 4, h2);
        printhash(h2);
    */

#ifdef RUST_POSEIDON
    ext_poseidon_hash_or_noop(h1, leaf, 6);
    printhash(h1);
#endif

    return 1;
}

int test2()
{
    // 4 leaves of 7 elements each -> Merkle tree has 7 nodes
    u64 test_leaves[28] = {
        12382199520291307008, 18193113598248284716, 17339479877015319223, 10837159358996869336, 9988531527727040483, 5682487500867411209, 13124187887292514366,
        8395359103262935841, 1377884553022145855, 2370707998790318766, 3651132590097252162, 1141848076261006345, 12736915248278257710, 9898074228282442027,
        10465118329878758468, 5866464242232862106, 15506463679657361352, 18404485636523119190, 15311871720566825080, 5967980567132965479, 14180845406393061616,
        15480539652174185186, 5454640537573844893, 3664852224809466446, 5547792914986991141, 5885254103823722535, 6014567676786509263, 11767239063322171808};

    // CPU
    u64 tree1[28] = {0ul};

    for (u32 i = 0; i < 4; i++)
    {
        cpu_poseidon_hash_one(test_leaves + 7 * i, 7, tree1 + 4 * i);
    }
    cpu_poseidon_hash_two(tree1, tree1 + 4, tree1 + 16);
    cpu_poseidon_hash_two(tree1 + 8, tree1 + 12, tree1 + 20);
    cpu_poseidon_hash_two(tree1 + 16, tree1 + 20, tree1 + 24);

    // GPU
    u64 tree2[28] = {0ul};

    u64 *gpu_leaf;
    u64 *gpu_hash;
    CHECKCUDAERR(hipMalloc(&gpu_leaf, 28 * sizeof(u64)));
    CHECKCUDAERR(hipMalloc(&gpu_hash, 28 * sizeof(u64)));
    CHECKCUDAERR(hipMemcpy(gpu_leaf, test_leaves, 28 * sizeof(u64), hipMemcpyHostToDevice));
    hash_step1<<<1, 4>>>(gpu_leaf, gpu_hash, 7, 4);
    hash_step2<<<1, 2>>>(gpu_hash, gpu_hash + 16, 2);
    hash_step2<<<1, 2>>>(gpu_hash + 16, gpu_hash + 24, 1);
    CHECKCUDAERR(hipMemcpy(tree2, gpu_hash, 28 * sizeof(u64), hipMemcpyDeviceToHost));

    int ret = 1;
    for (u32 i = 0; i < 28; i++)
    {
        if (tree1[i] != tree2[i])
        {
            printf("Diff at idx %u: %lu %lu\n", i, tree1[i], tree2[i]);
            ret = 0;
        }
    }
    if (ret == 1)
    {
        printf("Trees are the same!\n");
    }

    return ret;
}

int main()
{

    test1();

    test2();

    return 0;
}