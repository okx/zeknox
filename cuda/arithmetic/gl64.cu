#include "hip/hip_runtime.h"
#ifndef __CRYPTO_ARITHMEETIC_GL64_CU__
#define __CRYPTO_ARITHMEETIC_CL64_CU__
#include <arithmetic/kernels/gl64_kernel.cu>
#ifndef __CUDA_ARCH__   // below is cpu code; __CUDA_ARCH__ should not be defined

#if defined(EXPOSE_C_INTERFACE)
extern "C" 
#endif 
void goldilocks_add(fr_t *result, fr_t *a, fr_t *b)
{

    fr_t *d_result, *d_a, *d_b;
    hipMalloc((fr_t**)&d_result, sizeof(fr_t));
    hipMalloc((fr_t**)&d_a, sizeof(fr_t));
    hipMalloc((fr_t**)&d_b, sizeof(fr_t));

    hipMemcpy(d_a, a, sizeof(fr_t), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(fr_t), hipMemcpyHostToDevice);
    goldilocks_add_kernel<<<1,1>>>(
        d_result, d_a, d_b
        );

    hipMemcpy(result, d_result, sizeof(fr_t), hipMemcpyDeviceToHost);

}

#if defined(EXPOSE_C_INTERFACE)
extern "C" 
#endif 
void goldilocks_sub(fr_t *result, fr_t *a, fr_t *b)
{

    fr_t *d_result, *d_a, *d_b;
    hipMalloc((fr_t**)&d_result, sizeof(fr_t));
    hipMalloc((fr_t**)&d_a, sizeof(fr_t));
    hipMalloc((fr_t**)&d_b, sizeof(fr_t));

    hipMemcpy(d_a, a, sizeof(fr_t), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(fr_t), hipMemcpyHostToDevice);
    goldilocks_sub_kernel<<<1,1>>>(
        d_result, d_a, d_b
        );

    hipMemcpy(result, d_result, sizeof(fr_t), hipMemcpyDeviceToHost);

}

#if defined(EXPOSE_C_INTERFACE)
extern "C" 
#endif 
void goldilocks_mul(fr_t *result, fr_t *a, fr_t *b)
{
       fr_t *d_result, *d_a, *d_b;
    hipMalloc((fr_t**)&d_result, sizeof(fr_t));
    hipMalloc((fr_t**)&d_a, sizeof(fr_t));
    hipMalloc((fr_t**)&d_b, sizeof(fr_t));

    hipMemcpy(d_a, a, sizeof(fr_t), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(fr_t), hipMemcpyHostToDevice);
    goldilocks_mul_kernel<<<1,1>>>(
        d_result, d_a, d_b
        );

    hipMemcpy(result, d_result, sizeof(fr_t), hipMemcpyDeviceToHost);
}

#if defined(EXPOSE_C_INTERFACE)
extern "C" 
#endif 
void goldilocks_rshift(fr_t *result, fr_t *a, uint32_t *r)
{
       fr_t *d_result, *d_a;
       uint32_t *d_r;
    hipMalloc((fr_t**)&d_result, sizeof(fr_t));
    hipMalloc((fr_t**)&d_a, sizeof(fr_t));
    hipMalloc((uint32_t**)&d_r, sizeof(uint32_t));

    hipMemcpy(d_a, a, sizeof(fr_t), hipMemcpyHostToDevice);
    hipMemcpy(d_r, r, sizeof(uint32_t), hipMemcpyHostToDevice);
    goldilocks_rshift_kernel<<<1,1>>>(
        d_result, d_a, d_r
        );

    hipMemcpy(result, d_result, sizeof(fr_t), hipMemcpyDeviceToHost);
}


#if defined(EXPOSE_C_INTERFACE)
extern "C" 
#endif 
void goldilocks_inverse(fr_t *result, fr_t *a)
{
       fr_t *d_result, *d_a;
    hipMalloc((fr_t**)&d_result, sizeof(fr_t));
    hipMalloc((fr_t**)&d_a, sizeof(fr_t));

    hipMemcpy(d_a, a, sizeof(fr_t), hipMemcpyHostToDevice);
    goldilocks_inverse_kernel<<<1,1>>>(
        d_result, d_a
        );

    hipMemcpy(result, d_result, sizeof(fr_t), hipMemcpyDeviceToHost);
}

#if defined(EXPOSE_C_INTERFACE)
extern "C" 
#endif 
void goldilocks_exp(fr_t *result, fr_t *base, uint32_t *pow)
{
    fr_t *d_result, *d_base;
    uint32_t *d_pow;
    hipMalloc((fr_t **)&d_result, sizeof(fr_t));
    hipMalloc((fr_t **)&d_base, sizeof(fr_t));
    hipMalloc((uint32_t **)&d_pow, sizeof(uint32_t));

    hipMemcpy(d_base, base, sizeof(fr_t), hipMemcpyHostToDevice);
    hipMemcpy(d_pow, pow, sizeof(uint32_t), hipMemcpyHostToDevice);
    goldilocks_exp_kernel<<<1, 1>>>(
        d_result, d_base, d_pow);

    hipMemcpy(result, d_result, sizeof(fr_t), hipMemcpyDeviceToHost);
}
#endif
#endif