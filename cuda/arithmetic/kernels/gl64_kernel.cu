#include "hip/hip_runtime.h"
#ifndef __CRYPTO_ARITHMATIC_KERNELS_GL64_CU__
#define __CRYPTO_ARITHMATIC_KERNELS_GL64_CU__

#include <ff/goldilocks.hpp>
__global__ void goldilocks_add_kernel(
    fr_t *d_result, fr_t *d_a, fr_t *d_b)
{

    *d_result = *d_a + *d_b;
}

__global__ void goldilocks_sub_kernel(
    fr_t *d_result, fr_t *d_a, fr_t *d_b)
{

    *d_result = *d_a - *d_b;
}

__global__ void goldilocks_mul_kernel(fr_t *d_result, fr_t *d_a, fr_t *d_b)
{
    *d_result = *d_a * *d_b;
}

__global__ void goldilocks_inverse_kernel(fr_t *d_result, fr_t *d_a)
{
    *d_result = 1 / *d_a;
}

__global__ void goldilocks_rshift_kernel(fr_t *d_result, fr_t *d_a, uint32_t *r)
{
    *d_result = *d_a >> (*r);
}

__global__ void goldilocks_exp_kernel(fr_t *d_result, fr_t *d_a, uint32_t *r)
{
    *d_result = (*d_a) ^ (*r);
}
#endif