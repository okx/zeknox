#ifndef __CRYPTO_ARITHMEETIC_GL64_CU__
#define __CRYPTO_ARITHMEETIC_CL64_CU__
#include <arithmetic/kernels/bn254_kernel.cu>
#ifndef __CUDA_ARCH__   // below is cpu code; __CUDA_ARCH__ should not be defined

#if defined(EXPOSE_C_INTERFACE)
extern "C" 
#endif
void bn128_add(fp_t *result, fp_t *a, fp_t *b)
{
    fp_t *d_result, *d_a, *d_b;
    hipMalloc((fp_t**)&d_result, sizeof(fp_t));
    hipMalloc((fp_t**)&d_a, sizeof(fp_t));
    hipMalloc((fp_t**)&d_b, sizeof(fp_t));

    hipMemcpy(d_a, a, sizeof(fp_t), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(fp_t), hipMemcpyHostToDevice);
    bn128_add_kernel<<<1,1>>>(
        d_result, d_a, d_b
        );

    hipMemcpy(result, d_result, sizeof(fp_t), hipMemcpyDeviceToHost);

}
#if defined(EXPOSE_C_INTERFACE)
extern "C" 
#endif
void bn128_sub(fp_t *result, fp_t *a, fp_t *b)
{

    fp_t *d_result, *d_a, *d_b;
    hipMalloc((fp_t**)&d_result, sizeof(fp_t));
    hipMalloc((fp_t**)&d_a, sizeof(fp_t));
    hipMalloc((fp_t**)&d_b, sizeof(fp_t));

    hipMemcpy(d_a, a, sizeof(fp_t), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(fp_t), hipMemcpyHostToDevice);
    bn128_sub_kernel<<<1,1>>>(
        d_result, d_a, d_b
        );

    hipMemcpy(result, d_result, sizeof(fp_t), hipMemcpyDeviceToHost);

}

#if defined(EXPOSE_C_INTERFACE)
extern "C" 
#endif 
void bn128_mul(fp_t *result, fp_t *a, fp_t *b)
{

    fp_t *d_result, *d_a, *d_b;
    hipMalloc((fp_t**)&d_result, sizeof(fp_t));
    hipMalloc((fp_t**)&d_a, sizeof(fp_t));
    hipMalloc((fp_t**)&d_b, sizeof(fp_t));

    hipMemcpy(d_a, a, sizeof(fp_t), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(fp_t), hipMemcpyHostToDevice);
    bn128_mul_kernel<<<1,1>>>(
        d_result, d_a, d_b
        );

    hipMemcpy(result, d_result, sizeof(fp_t), hipMemcpyDeviceToHost);

}

#if defined(EXPOSE_C_INTERFACE)
extern "C" 
#endif 
void bn128_lshift(fp_t *result, fp_t *a, uint32_t *r)
{
       fp_t *d_result, *d_a;
       uint32_t *d_r;
    hipMalloc((fp_t**)&d_result, sizeof(fp_t));
    hipMalloc((fp_t**)&d_a, sizeof(fp_t));
    hipMalloc((uint32_t**)&d_r, sizeof(uint32_t));

    hipMemcpy(d_a, a, sizeof(fp_t), hipMemcpyHostToDevice);
    hipMemcpy(d_r, r, sizeof(uint32_t), hipMemcpyHostToDevice);
    bn128_lshift_kernel<<<1,1>>>(
        d_result, d_a, d_r
        );

    hipMemcpy(result, d_result, sizeof(fp_t), hipMemcpyDeviceToHost);
}


#if defined(EXPOSE_C_INTERFACE)
extern "C" 
#endif 
void bn128_rshift(fp_t *result, fp_t *a, uint32_t *r)
{
       fp_t *d_result, *d_a;
       uint32_t *d_r;
    hipMalloc((fp_t**)&d_result, sizeof(fp_t));
    hipMalloc((fp_t**)&d_a, sizeof(fp_t));
    hipMalloc((uint32_t**)&d_r, sizeof(uint32_t));

    hipMemcpy(d_a, a, sizeof(fp_t), hipMemcpyHostToDevice);
    hipMemcpy(d_r, r, sizeof(uint32_t), hipMemcpyHostToDevice);
    bn128_rshift_kernel<<<1,1>>>(
        d_result, d_a, d_r
        );

    hipMemcpy(result, d_result, sizeof(fp_t), hipMemcpyDeviceToHost);
}

#endif
#endif