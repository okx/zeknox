#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#ifndef __DEBUG__PRINT__
#define __DEBUG__PRINT__
#include <cstdio>
#endif

#if defined(FEATURE_GOLDILOCKS)

#include <ff/goldilocks.hpp>
#elif defined(FEATURE_BN128)
# include <ff/alt_bn128.hpp>
#else
#error "no FEATURE"
#endif

void __global__ print()
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    std::printf("%d\n", idx);
}

void print_function()
{
    print<<<1, 10>>>();
    hipDeviceSynchronize();
}

#include <util/cuda_available.hpp>
#include <ntt/ntt.cuh>
#include <arithmetic/arithmetic.hpp>
#ifndef __CUDA_ARCH__   // below is cpu code; __CUDA_ARCH__ should not be defined


extern "C" RustError compute_ntt(size_t device_id, fr_t *inout, uint32_t lg_domain_size,
                                 NTT::InputOutputOrder ntt_order,
                                 NTT::Direction ntt_direction,
                                 NTT::Type ntt_type)
{
    auto &gpu = select_gpu(device_id);

    return NTT::Base(gpu, inout, lg_domain_size,
                     ntt_order, ntt_direction, ntt_type);
}
#endif

