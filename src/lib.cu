#include <hip/hip_runtime.h>
#ifndef __DEBUG__PRINT__
#define __DEBUG__PRINT__
#include <cstdio>
#endif
#if defined(FEATURE_GOLDILOCKS)
#include <ff/goldilocks.hpp>
#include <ff/arithmatic.cuh>
#else
#error "no FEATURE"
#endif
#include <util/cuda_available.hpp>
#include <ntt/ntt.cuh>

#ifndef __CUDA_ARCH__   // below is cpu code; __CUDA_ARCH__ should not be defined

extern "C" void goldilocks_add(fr_t *result, fr_t *a, fr_t *b)
{

    fr_t *d_result, *d_a, *d_b;
    hipMalloc((fr_t**)&d_result, sizeof(fr_t));
    hipMalloc((fr_t**)&d_a, sizeof(fr_t));
    hipMalloc((fr_t**)&d_b, sizeof(fr_t));

    hipMemcpy(d_a, a, sizeof(fr_t), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(fr_t), hipMemcpyHostToDevice);
    goldilocks_add_kernel<<<1,1>>>(
        d_result, d_a, d_b
        );

    hipMemcpy(result, d_result, sizeof(fr_t), hipMemcpyDeviceToHost);

}

extern "C" void goldilocks_sub(fr_t *result, fr_t *a, fr_t *b)
{

    fr_t *d_result, *d_a, *d_b;
    hipMalloc((fr_t**)&d_result, sizeof(fr_t));
    hipMalloc((fr_t**)&d_a, sizeof(fr_t));
    hipMalloc((fr_t**)&d_b, sizeof(fr_t));

    hipMemcpy(d_a, a, sizeof(fr_t), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(fr_t), hipMemcpyHostToDevice);
    goldilocks_sub_kernel<<<1,1>>>(
        d_result, d_a, d_b
        );

    hipMemcpy(result, d_result, sizeof(fr_t), hipMemcpyDeviceToHost);

}

extern "C" void goldilocks_mul(fr_t *result, fr_t *a, fr_t *b)
{
       fr_t *d_result, *d_a, *d_b;
    hipMalloc((fr_t**)&d_result, sizeof(fr_t));
    hipMalloc((fr_t**)&d_a, sizeof(fr_t));
    hipMalloc((fr_t**)&d_b, sizeof(fr_t));

    hipMemcpy(d_a, a, sizeof(fr_t), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(fr_t), hipMemcpyHostToDevice);
    goldilocks_mul_kernel<<<1,1>>>(
        d_result, d_a, d_b
        );

    hipMemcpy(result, d_result, sizeof(fr_t), hipMemcpyDeviceToHost);
}

extern "C" void goldilocks_rshift(fr_t *result, fr_t *a, uint32_t *r)
{
       fr_t *d_result, *d_a;
       uint32_t *d_r;
    hipMalloc((fr_t**)&d_result, sizeof(fr_t));
    hipMalloc((fr_t**)&d_a, sizeof(fr_t));
    hipMalloc((uint32_t**)&d_r, sizeof(uint32_t));

    hipMemcpy(d_a, a, sizeof(fr_t), hipMemcpyHostToDevice);
    hipMemcpy(d_r, r, sizeof(uint32_t), hipMemcpyHostToDevice);
    goldilocks_rshift_kernel<<<1,1>>>(
        d_result, d_a, d_r
        );

    hipMemcpy(result, d_result, sizeof(fr_t), hipMemcpyDeviceToHost);
}


extern "C" void goldilocks_inverse(fr_t *result, fr_t *a)
{
       fr_t *d_result, *d_a;
    hipMalloc((fr_t**)&d_result, sizeof(fr_t));
    hipMalloc((fr_t**)&d_a, sizeof(fr_t));

    hipMemcpy(d_a, a, sizeof(fr_t), hipMemcpyHostToDevice);
    goldilocks_inverse_kernel<<<1,1>>>(
        d_result, d_a
        );

    hipMemcpy(result, d_result, sizeof(fr_t), hipMemcpyDeviceToHost);
}
#endif
