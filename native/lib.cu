// Copyright 2024 OKX Group
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

/** macro utils*/
#define XSTR(x) STR(x)
#define STR(x) #x

/*
#ifdef NDEBUG
#define CUDA_DEBUG false
#else
#define CUDA_DEBUG true
#include <cstdio>
#endif
*/

// #pragma message "The value of BUILD_MSM: " XSTR(BUILD_MSM)
// #pragma message "The value of __CUDA_ARCH__: " XSTR(__CUDA_ARCH__)

#include <hip/hip_runtime.h>
#include <utils/gpu_t.cuh>
#include <utils/all_gpus.hpp>
#include <ntt/ntt.cuh>
#include "lib.h"

#if defined(FEATURE_GOLDILOCKS)
#include <ff/goldilocks.hpp>
#elif defined(FEATURE_BN254)
#include <ff/alt_bn254.hpp>
#else
#error "no FEATURE"
#endif
#include <utils/device_context.cuh>

#if defined(EXPOSE_C_INTERFACE)
extern "C"
#endif
    RustError
    list_devices_info()
{
    list_all_gpus_prop();
    return RustError{hipSuccess};
}

#if defined(FEATURE_GOLDILOCKS)
#include <ff/goldilocks.hpp>
#elif defined(FEATURE_BN254)
#include <ff/alt_bn254.hpp>
#else
#error "no FEATURE"
#endif

#include <ntt/ntt.h>
#ifdef BUILD_MSM
#include <msm/msm.h>
#endif
#include <vector>

#ifndef __CUDA_ARCH__ // below is cpu code; __CUDA_ARCH__ should not be defined

#if defined(EXPOSE_C_INTERFACE)
extern "C"
#endif
    RustError
    get_number_of_gpus(size_t *nums)
{
    *nums = ngpus();
    return RustError{hipSuccess};
}

#if defined(EXPOSE_C_INTERFACE)
extern "C"
#endif
    RustError
    compute_batched_ntt(size_t device_id, void *inout, uint32_t lg_domain_size,
                        NTT_Direction ntt_direction, NTT_Config cfg)
{
    auto &gpu = select_gpu(device_id);
    return ntt::batch_ntt(gpu, (fr_t *)inout, lg_domain_size, ntt_direction, cfg);
}

#if defined(EXPOSE_C_INTERFACE)
extern "C"
#endif
    RustError
    compute_batched_lde(size_t device_id, void *output, void *input, uint32_t lg_domain_size,
                        NTT_Direction ntt_direction, NTT_Config cfg)
{
    auto &gpu = select_gpu(device_id);
    return ntt::batch_lde(gpu, (fr_t *)output, (fr_t *)input, lg_domain_size, ntt_direction, cfg);
}

#if defined(EXPOSE_C_INTERFACE)
extern "C"
#endif
    RustError
    compute_batched_lde_multi_gpu(void *output, void *input, uint32_t num_gpu, NTT_Direction ntt_direction,
                                  NTT_Config cfg, uint32_t lg_domain_size, size_t total_num_input_elements, size_t total_num_output_elements)
{
    return ntt::batch_lde_multi_gpu((fr_t *)output, (fr_t *)input, num_gpu, ntt_direction, cfg, lg_domain_size, total_num_input_elements, total_num_output_elements);
}

#if defined(EXPOSE_C_INTERFACE)
extern "C"
#endif
    RustError
    compute_transpose_rev(size_t device_id, void *output, void *input, uint32_t lg_n,
                          NTT_TransposeConfig cfg)
{
    auto &gpu = select_gpu(device_id);
    return ntt::compute_transpose_rev(gpu, (fr_t*)output, (fr_t*)input, lg_n, cfg);
}

#if defined(EXPOSE_C_INTERFACE)
extern "C"
#endif
    RustError
    init_twiddle_factors(size_t device_id, size_t lg_n)
{
    auto &gpu = select_gpu(device_id);
    return ntt::init_twiddle_factors(gpu, lg_n);
}

#if defined(EXPOSE_C_INTERFACE)
extern "C"
#endif
    RustError
    init_coset(size_t device_id, size_t lg_n, const uint64_t coset_gen)
{
    auto &gpu = select_gpu(device_id);
    return ntt::init_coset(gpu, lg_n, fr_t(coset_gen));
}

#endif

#if defined(EXPOSE_C_INTERFACE)
extern "C"
#endif
    void
    init_cuda_degree_and_generator(const uint32_t max_degree, const uint64_t group_generator)
{
#if defined(FEATURE_GOLDILOCKS)
    size_t num_of_gpus = ngpus();

    for (size_t d = 0; d < num_of_gpus; d++)
    {
        init_coset(d, max_degree, group_generator);
        for (size_t k = 2; k <= max_degree; k++)
        {
            init_twiddle_factors(d, k);
        }
    }
#endif
}

#if defined(EXPOSE_C_INTERFACE)
extern "C"
#endif
    void
    init_cuda_degree(const uint32_t max_degree)
{
#if defined(FEATURE_GOLDILOCKS)
    init_cuda_degree_and_generator(max_degree, GROUP_GENERATOR);
#endif
}

#if defined(EXPOSE_C_INTERFACE)
extern "C"
#endif
    void
    init_cuda()
{
    init_cuda_degree(24);
}