// Copyright 2024 OKX
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

/** macro utils*/
#define XSTR(x) STR(x)
#define STR(x) #x

/*
#ifdef NDEBUG
#define CUDA_DEBUG false
#else
#define CUDA_DEBUG true
#include <cstdio>
#endif
*/

// #pragma message "The value of CUDA_DEBUG: " XSTR(CUDA_DEBUG)
// #pragma message "The value of __CUDA_ARCH__: " XSTR(__CUDA_ARCH__)

#include <hip/hip_runtime.h>
#include <utils/gpu_t.cuh>
#include <utils/cuda_available.hpp>
#include <utils/all_gpus.cpp>
#include "lib.h"


#if defined(EXPOSE_C_INTERFACE)
extern "C"
#endif
RustError list_devices_info(){
    list_all_gpus_prop();
    return RustError{hipSuccess};
}

#if defined(FEATURE_GOLDILOCKS)
#include <ff/goldilocks.hpp>
#elif defined(FEATURE_BN254)
#include <ff/alt_bn254.hpp>
#else
#error "no FEATURE"
#endif

#include <ntt/ntt.cuh>
#include <ntt/ntt.h>
#include <vector>



#ifndef __CUDA_ARCH__ // below is cpu code; __CUDA_ARCH__ should not be defined

#if defined(EXPOSE_C_INTERFACE)
extern "C"
#endif
    RustError
    get_number_of_gpus(size_t *nums) {
    *nums = ngpus();
    return RustError{hipSuccess};
}


#if defined(EXPOSE_C_INTERFACE)
extern "C"
#endif
    RustError
    compute_batched_ntt(size_t device_id, fr_t *inout, uint32_t lg_domain_size,
                        Ntt_Types::Direction ntt_direction, Ntt_Types::NTTConfig cfg)
{
    auto &gpu = select_gpu(device_id);
    return ntt::batch_ntt(gpu, inout, lg_domain_size, ntt_direction, cfg);
}

#if defined(EXPOSE_C_INTERFACE)
extern "C"
#endif
RustError
compute_batched_lde(size_t device_id, fr_t *output, fr_t *input, uint32_t lg_domain_size,
                        Ntt_Types::Direction ntt_direction, Ntt_Types::NTTConfig cfg)
{
    auto &gpu = select_gpu(device_id);
    return ntt::batch_lde(gpu, output, input, lg_domain_size, ntt_direction, cfg);
}

#if defined(EXPOSE_C_INTERFACE)
extern "C"
#endif
RustError
compute_batched_lde_multi_gpu(fr_t *output,fr_t *input, uint32_t num_gpu, Ntt_Types::Direction ntt_direction,
                        Ntt_Types::NTTConfig cfg, uint32_t lg_domain_size, size_t total_num_input_elements, size_t total_num_output_elements)
{
    return ntt::batch_lde_multi_gpu(output, input, num_gpu, ntt_direction, cfg, lg_domain_size, total_num_input_elements, total_num_output_elements);
}

#if defined(EXPOSE_C_INTERFACE)
extern "C"
#endif
RustError
compute_transpose_rev(size_t device_id, fr_t *output, fr_t *input, uint32_t lg_n,
                        Ntt_Types::TransposeConfig cfg) {
    auto &gpu = select_gpu(device_id);
    return ntt::compute_transpose_rev(gpu, output, input, lg_n, cfg);
}

#if defined(EXPOSE_C_INTERFACE)
extern "C"
#endif
RustError
compute_naive_transpose_rev(size_t device_id, fr_t *output, fr_t *input, uint32_t lg_n,
                        Ntt_Types::TransposeConfig cfg)
{
    auto &gpu = select_gpu(device_id);
    return ntt::compute_naive_transpose_rev(gpu, output, input, lg_n, cfg);
}

#if defined(EXPOSE_C_INTERFACE)
extern "C"
#endif
    RustError
    init_twiddle_factors(size_t device_id, size_t lg_n)
{
    auto &gpu = select_gpu(device_id);
    return ntt::init_twiddle_factors(gpu, lg_n);
}

#if defined(EXPOSE_C_INTERFACE)
extern "C"
#endif
    RustError
    init_coset(size_t device_id, size_t lg_n, fr_t coset_gen)
{
    auto &gpu = select_gpu(device_id);
    return ntt::init_coset(gpu, lg_n, coset_gen);
}

#endif

#ifndef FEATURE_GOLDILOCKS
#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>

#include <msm/pippenger.cuh>
#include <cstdio>
#include <blst_t.hpp>

RustError::by_value mult_pippenger(point_t *result, const affine_t points[],
                                   size_t npoints, const scalar_t scalars[],
                                   size_t ffi_affine_sz) {
    RustError r = mult_pippenger<bucket_t>(result, points, npoints, scalars, false, ffi_affine_sz);
    return r;
}

#if defined(G2_ENABLED)
extern "C" RustError::by_value mult_pippenger_g2(g2_projective_t *result, g2_affine_t *points, size_t msm_size, scalar_field_t *scalars, size_t large_bucket_factor, bool on_device, bool big_triangle)
{
    mult_pippenger_g2_internal<scalar_field_t, g2_projective_t, g2_affine_t>(
        result, points, scalars, msm_size, on_device, big_triangle, large_bucket_factor);
    CHECK_LAST_CUDA_ERROR();
}
#endif
#endif

#if defined(EXPOSE_C_INTERFACE)
extern "C"
#endif
void init_cuda() {
    // This is taken from Plonky2 field (MULTIPLICATIVE_GROUP_GENERATOR = 7)
#if defined(FEATURE_GOLDILOCKS)
    const fr_t generator = fr_t(7);

    size_t num_of_gpus = ngpus();

    for (size_t d = 0; d < num_of_gpus; d++)
    {
        init_coset(d, 24, generator);
        for (size_t k = 2; k < 25; k++)
        {
            init_twiddle_factors(d, k);
        }
    }
#endif
}

#if defined(EXPOSE_C_INTERFACE)
extern "C"
#endif
void init_cuda_degree(uint32_t max_degree) {
    // This is taken from Plonky2 field (MULTIPLICATIVE_GROUP_GENERATOR = 7)
#if defined(FEATURE_GOLDILOCKS)
    const fr_t generator = fr_t(7);

    size_t num_of_gpus = ngpus();

    for (size_t d = 0; d < num_of_gpus; d++)
    {
        init_coset(d, max_degree, generator);
        for (size_t k = 2; k <= max_degree; k++)
        {
            init_twiddle_factors(d, k);
        }
    }
#endif
}